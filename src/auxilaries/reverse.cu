#include "hip/hip_runtime.h"
float **__autogen_device_main_c;
short __autogen_device_main_c_prstn = 0;
float **__autogen_device_main_a;
short __autogen_device_main_a_prstn = 0;
float **__autogen_device_main_b;
short __autogen_device_main_b_prstn = 0;

#include <malloc.h>
#include <time.h>
#include <openacc.h>

#include <math.h>

#define SIZE 1000
int
main ()
{
  int i, j;

  float a[SIZE][SIZE];
  float b[SIZE][SIZE];
  float c[SIZE][SIZE];
  float seq[SIZE][SIZE];
  acc_init (acc_device_nvidia);



  for (i = 0; i < SIZE; ++i)
    {

      for (j = 0; j < SIZE; ++j)
	{

	  a[i][j] = (float) i + j;
	  b[i][j] = (float) i - j;
	  c[i][j] = 0.0f;
	}
    }

  unsigned long long int tic, toc;

  int k;
  for (k = 0; k < 3; k++)
    {

      printf ("Calculation on GPU ... ");
      tic = clock ();


      if (!__autogen_device_main_c_prstn)
	{
	  __autogen_device_main_c_prstn++;
	  hipMalloc ((void **) &c, (1000) * (1000) * sizeof (float));
	}
      hipMemcpy (__autogen_device_main_c, c,
		  (1000) * (1000) * sizeof (float), hipMemcpyHostToDevice);
      if (!__autogen_device_main_a_prstn)
	{
	  __autogen_device_main_a_prstn++;
	  hipMalloc ((void **) &a, (1000) * (1000) * sizeof (float));
	}
      hipMemcpy (__autogen_device_main_a, a,
		  (1000) * (1000) * sizeof (float), hipMemcpyHostToDevice);
      if (!__autogen_device_main_b_prstn)
	{
	  __autogen_device_main_b_prstn++;
	  hipMalloc ((void **) &b, (1000) * (1000) * sizeof (float));
	}
      hipMemcpy (__autogen_device_main_b, b,
		  (1000) * (1000) * sizeof (float), hipMemcpyHostToDevice);


      {

	__ungenerated_kernel_region_0 ();

      }
      hipMemcpy (c, __autogen_device_main_c,
		  (1000) * (1000) * sizeof (float), hipMemcpyDeviceToHost);


      toc = clock ();
      printf (" %6.4f ms\n", (toc - tic) / (float) 1000);
    }





  printf ("Calculation on CPU ... ");
  tic = clock ();
  for (i = 0; i < SIZE; ++i)
    {

      for (j = 0; j < SIZE; ++j)
	{

	  seq[i][j] = sin (a[i][j]) + cos (b[i][j]) + cos (a[i][j] * b[i][j]);
	  if (c[i][j] != seq[i][j])
	    {
	      printf ("Error %d %d\n", i, j);
	      exit (1);
	    }
	}
    }
  toc = clock ();
  printf (" %6.4f ms\n", (toc - tic) / (float) 1000);

  printf ("OpenACC vector add test was successful!\n");

  return 0;
}
